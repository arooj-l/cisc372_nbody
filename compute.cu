#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include <hip/hip_runtime.h>
#include "compute.h"
#include <hip/hip_runtime.h>


#define BLOCKSIZE 256


// Device function, to compute the pairwise acceleration between objects
__device__ void computeAcceleration(const vector3* __restrict__ pos, const double* __restrict__ mass,  vector3* __restrict__ accels)
{
  
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

   
    if (tid >= NUMENTITIES)
        return;

    for (int i = 0; i < NUMENTITIES; i++)
    {
        if (i == tid)
        {
            accels[tid][0] = 0.0;
            accels[tid][1] = 0.0;
            accels[tid][2] = 0.0;
        }
        else
        {
            vector3 distance;
            distance[0] = pos[tid][0] - pos[i][0];
            distance[1] = pos[tid][1] - pos[i][1];
            distance[2] = pos[tid][2] - pos[i][2];

            double magnitude_sq = distance[0] * distance[0] + distance[1] * distance[1] + distance[2] * distance[2];
            double magnitude = sqrt(magnitude_sq);
            double accelmag = -1.0 * GRAV_CONSTANT * mass[i] / magnitude_sq;

            accels[tid][0] += accelmag * distance[0] / magnitude;
            accels[tid][1] += accelmag * distance[1] / magnitude;
            accels[tid][2] += accelmag * distance[2] / magnitude;
        }
    }
}

// Kernel function
__global__ void compute(vector3* __restrict__ pos, vector3* __restrict__ vel, const double* __restrict__ mass)
{
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

   
    if (tid >= NUMENTITIES)
        return;


    vector3 accels[NUMENTITIES];

   
    computeAcceleration(pos, mass, accels);

    
    double ax = accels[tid][0];
    double ay = accels[tid][1];
    double az = accels[tid][2];

   
    vel[tid][0] += ax * INTERVAL;
    vel[tid][1] += ay * INTERVAL;
    vel[tid][2] += az * INTERVAL;

    pos[tid][0] += vel[tid][0] * INTERVAL;
    pos[tid][1] += vel[tid][1] * INTERVAL;
    pos[tid][2] += vel[tid][2] * INTERVAL;
}
